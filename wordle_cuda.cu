#include "hip/hip_runtime.h"
#include "wordle.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#define SIZE_WORDS 5

char *dictionary;
char *dictionaryGPU;
int tamDictionary;
int sizeWords;

__global__ void calculateEntropiesGPU(entropyType *, char *, int, int);
__device__ double calculateSingleEntropyGPU(char *, int, int, char *, char *, int);
__device__ int calculateProbGPU(char *, int, int, char *, char *);

/*
  The main function is responsible for checking that the number of parameters are correct, that the file can be 
  opened and for load the dictionary from the file
*/
int main(int argc, char **argv) {
    if (argc < 3) {
        fprintf(stderr, "Length of the words and file name with all words of dictionary must be specified in the first argument\n");
        exit(ERROR_ARGUMENTS);
    }
    else if (argc > 3) {
        fprintf(stderr, "Incorrect number of arguments, it must be only two\n");
        exit(ERROR_ARGUMENTS);
    }

    FILE *file;
    if ((file = fopen(argv[2], "r")) == NULL) {
        fprintf(stderr, "File \"%s\" cannot be opened\n", argv[2]);
        exit(ERROR_FILE_OPENING);
    }

    sizeWords = atoi(argv[1]);

    char line[sizeWords + 2];
    while(fgets(line, sizeWords + 2, file) != NULL) {
        if (tamDictionary % 64 == 0) {
            if ((dictionary = (char *) realloc(dictionary, sizeWords*(tamDictionary + 64))) == NULL) {
                fprintf(stderr, "Dynamic memory couldn't be allocated\n");
                exit(ERROR_DYNAMIC_MEMORY_ALLOCATION);
            }
        }

        for (int i = 0; i < sizeWords; i++) {
            if (line[i] == '\n') {
                fprintf(stderr, "Length of word \"%s\" at line %d does not match the specified length of the words\n", line, tamDictionary + 1);
                exit(ERROR_LENGTH_WORD);
            }

            dictionary[tamDictionary*sizeWords + i] = line[i];
        }

        tamDictionary++;
    }

    fclose(file);

    wordle();
}


/*
  The wordle function is the main function of the program, where an array of entropies are calculated and sorted to 
  give the 10 best options to the user
*/
void wordle() {
    int iteration = 1;

    //If iteration is 7 it means that the user lost the game
    while (iteration < 7) {
        //Entropies calculations and sorting
        entropyType entropies[tamDictionary];

        if (tamDictionary < 350) {
            calculateEntropies(entropies);
        }
        else {
            entropyType *entropiesGPU;

            hipMalloc(&entropiesGPU, tamDictionary*sizeof(entropyType));
            hipMalloc(&dictionaryGPU, tamDictionary*sizeWords);
            hipMemcpy(dictionaryGPU, dictionary, tamDictionary*sizeWords, hipMemcpyHostToDevice);

            int n_threads;
            int n_blocks;

            n_blocks = tamDictionary/768 + 1;
            if (tamDictionary < 768) {
                n_threads = tamDictionary;
            }
            else {
                n_threads = 768;
            }

            calculateEntropiesGPU<<<n_blocks, n_threads>>>(entropiesGPU, dictionaryGPU, tamDictionary, sizeWords);

            hipMemcpy(entropies, entropiesGPU, tamDictionary*sizeof(entropyType), hipMemcpyDeviceToHost);

            hipFree(dictionaryGPU);
            hipFree(entropiesGPU);
        }

        qsort(entropies, tamDictionary, sizeof(entropyType), compEntropies);

        //Giving the 10 best results to the user
        //If there isn't 10 options, it gives all the options
        for (int i = 0; i < 10; i++) {
            if (i == tamDictionary) {
                break;
            }
            char word[sizeWords + 1];
            strncpy(word, dictionary+entropies[i].position*sizeWords, sizeWords);
            word[sizeWords] = '\0';

            printf("%d: %s\n", i, word);
        }

        //Here the program ask the user which word has been selected and the result wordle gave
        char word[sizeWords];
        char colors[sizeWords];
        char useless;

        if (scanf("%c%c%c%c%c%c", word, word + 1, word + 2, word + 3, word + 4, &useless) < 6) {
            fprintf(stderr, "A word of length %d must be given\n", sizeWords);
            exit(ERROR_LENGTH_WORD);
        }
        if (scanf("%c%c%c%c%c%c", colors, colors + 1, colors + 2, colors + 3, colors + 4, &useless) < 6) {
            fprintf(stderr, "A color sequence of length %d must be given\n", sizeWords);
            exit(ERROR_LENGTH_COLOR);
        }

        for (int x = 0; x < sizeWords; x++) {
            if (colors[x] == 'g') {
                colors[x] = 'G';
                continue;
            }
            else if (colors[x] == 'y') {
                colors[x] = 'Y';
                continue;
            }
            else if (colors[x] == 'r') {
                colors[x] = 'R';
                continue;
            }
            if (colors[x] != 'G' && colors[x] != 'Y' && colors[x] != 'R') {
                fprintf(stderr, "Colors must be GYR\n");
                exit(ERROR_INVALID_COLOR);
            }
        }

        //Finally the dictionary is reduced following the word selected and color pattern obtained
        reduceDictionary(word, colors);

        if (tamDictionary == 0) {
            printf("No words are possible\n");
            exit(ERROR_NO_WORDS_POSSIBLE);
        }
    }
}


/*
  In this function all entropies are calculated
*/
void calculateEntropies(entropyType *entropies) {
    for (int i = 0; i < tamDictionary; i++) {
        char colors[sizeWords];

        entropies[i].position = i;
        entropies[i].entropy = -(calculateSingleEntropy(dictionary+i*sizeWords, colors, 0));
    }
}


/*
  Same function for gpu
*/
__global__ void calculateEntropiesGPU(entropyType *entropies, char *dictionary, int tamDictionary, int sizeWords) {
    char colors[SIZE_WORDS];

    int pos = blockIdx.x*blockDim.x + threadIdx.x;

    if(pos < tamDictionary) {
        entropies[pos].position = pos;
        entropies[pos].entropy = -(calculateSingleEntropyGPU(dictionary, tamDictionary, sizeWords, dictionary+pos*sizeWords, colors, 0));
    }
}


/*
  This is a recursive function that calculates the entropy of a word adding the information that a color pattern gives 
  to each other weighted
*/
double calculateSingleEntropy(char *word, char *colors, int depth) {
    double information = 0;

    if (depth == sizeWords - 1) {
        //In this case the color pattern is complete, so its information can be calculated

        colors[depth] = 'G';
        information += calculateProb(word, colors)/tamDictionary;
        if (information != 0) {
            information *= log2(information);
        }
        
        colors[depth] = 'Y';
        information += calculateProb(word, colors)/tamDictionary;
        if (information != 0) {
            information *= log2(information);
        }

        colors[depth] = 'R';
        information += calculateProb(word, colors)/tamDictionary;
        if (information != 0) {
            information *= log2(information);
        }
    }
    else {
        //In this case the color pattern isn't complete, so its information is calculated in the function it calls

        colors[depth] = 'G';
        information += calculateSingleEntropy(word, colors, depth + 1);
        
        colors[depth] = 'Y';
        information += calculateSingleEntropy(word, colors, depth + 1);

        colors[depth] = 'R';
        information += calculateSingleEntropy(word, colors, depth + 1);
    }

    return information;
}


/*
  Same function for gpu
*/
__device__ double calculateSingleEntropyGPU(char *dictionary, int tamDictionary, int sizeWords, char *word, char *colors, int depth) {
    double information = 0;

    if (depth == sizeWords - 1) {
        //In this case the color pattern is complete, so its information can be calculated

        colors[depth] = 'G';
        information += calculateProbGPU(dictionary, tamDictionary, sizeWords, word, colors)/tamDictionary;
        if (information != 0) {
            information *= log2(information);
        }
        
        colors[depth] = 'Y';
        information += calculateProbGPU(dictionary, tamDictionary, sizeWords, word, colors)/tamDictionary;
        if (information != 0) {
            information *= log2(information);
        }

        colors[depth] = 'R';
        information += calculateProbGPU(dictionary, tamDictionary, sizeWords, word, colors)/tamDictionary;
        if (information != 0) {
            information *= log2(information);
        }
    }
    else {
        //In this case the color pattern isn't complete, so its information is calculated in the function it calls

        colors[depth] = 'G';
        information += calculateSingleEntropyGPU(dictionary, tamDictionary, sizeWords, word, colors, depth + 1);
        
        colors[depth] = 'Y';
        information += calculateSingleEntropyGPU(dictionary, tamDictionary, sizeWords, word, colors, depth + 1);

        colors[depth] = 'R';
        information += calculateSingleEntropyGPU(dictionary, tamDictionary, sizeWords, word, colors, depth + 1);
    }

    return information;
}


/*
  This function calculates the number of words that follows the pattern of the colors and word given
*/
int calculateProb(char *word, char *colors) {
    int numWords = 0;

    /*
      For each letter of each word of the dictionary
        if green is read in word given and letters don't match, the word cannot be a candidate
        if yellow is read in word given and letters match, the word cannot be a candidate
          if letters don't match then it's added to yellow letters list to see if it exists in another position
        if red is read in word given and letters match, the word cannot be a candidate
          if letters don't match then it's added to red letters list to check that it isn't exist in another position
    */
    for (int i = 0; i < tamDictionary; i++) {
        int numYellow = 0;
        char yellows[sizeWords];
        int numRed = 0;
        char red[sizeWords];
        int possible = 0;
        char validate[sizeWords];
        int pos;

        for (int j = 0; j < sizeWords; j++) {
            pos = i*sizeWords + j;

            for (int x = 0; x < numRed; x++) {
                if (red[x] == dictionary[pos]) {
                    break;
                }
            }

            if (colors[j] == 'G') {
                if (word[j] != dictionary[pos]) {
                    break;
                }
                else {
                    validate[j] = 'Y';
                }
            }
            else if (colors[j] == 'Y') {
                if (word[j] == dictionary[pos]) {
                    break;
                }
                else {
                    yellows[numYellow] = word[j];

                    numYellow++;
                    validate[j] = 'N';
                }
            }
            else {
                if (word[j] == dictionary[pos]) {
                    break;
                }
                else {
                    red[numRed] = word[j];

                    numRed++;
                    validate[j] = 'N';
                }
            }

            possible++;
        }

        if (possible < sizeWords) {
            continue;
        }

        possible = 1;

        int ny = numYellow;
        for (int x = 0; x < sizeWords; x++) {
            for (int y = 0; y < numYellow; y++) {
                if (yellows[y] == dictionary[i*sizeWords + x]) {
                    yellows[y] = 0;
                    ny--;
                    validate[x] = 'Y';
                    break;
                }
            }

            if (validate[x] == 'N') {
                for (int y = 0; y < numRed; y++) {
                    if (red[y] == dictionary[i*sizeWords + x]) {
                        possible = 0;
                        break;
                    }
                }
            }

            if (!possible) {
                break;
            }
        }

        if (!possible || ny > 0) {
            continue;
        }

        numWords++;
    }

    return numWords;
}


/*
  Same function for gpu
*/
__device__ int calculateProbGPU(char *dictionary, int tamDictionary, int sizeWords, char *word, char *colors) {
    int numWords = 0;
    char yellows[SIZE_WORDS];
    char red[SIZE_WORDS];
    char validate[SIZE_WORDS];

    /*
      For each letter of each word of the dictionary
        if green is read in word given and letters don't match, the word cannot be a candidate
        if yellow is read in word given and letters match, the word cannot be a candidate
          if letters don't match then it's added to yellow letters list to see if it exists in another position
        if red is read in word given and letters match, the word cannot be a candidate
          if letters don't match then it's added to red letters list to check that it isn't exist in another position
    */
    for (int i = 0; i < tamDictionary; i++) {
        int numYellow = 0;
        int numRed = 0;
        int possible = 0;
        int pos;

        for (int j = 0; j < sizeWords; j++) {
            pos = i*sizeWords + j;

            for (int x = 0; x < numRed; x++) {
                if (red[x] == dictionary[pos]) {
                    break;
                }
            }

            if (colors[j] == 'G') {
                if (word[j] != dictionary[pos]) {
                    break;
                }
                else {
                    validate[j] = 'Y';
                }
            }
            else if (colors[j] == 'Y') {
                if (word[j] == dictionary[pos]) {
                    break;
                }
                else {
                    yellows[numYellow] = word[j];

                    numYellow++;
                    validate[j] = 'N';
                }
            }
            else {
                if (word[j] == dictionary[pos]) {
                    break;
                }
                else {
                    red[numRed] = word[j];

                    numRed++;
                    validate[j] = 'N';
                }
            }

            possible++;
        }

        if (possible < sizeWords) {
            continue;
        }

        possible = 1;

        int ny = numYellow;
        for (int x = 0; x < sizeWords; x++) {
            for (int y = 0; y < numYellow; y++) {
                if (yellows[y] == dictionary[i*sizeWords + x]) {
                    yellows[y] = 0;
                    ny--;
                    validate[x] = 'Y';
                    break;
                }
            }

            if (validate[x] == 'N') {
                for (int y = 0; y < numRed; y++) {
                    if (red[y] == dictionary[i*sizeWords + x]) {
                        possible = 0;
                        break;
                    }
                }
            }

            if (!possible) {
                break;
            }
        }

        if (!possible || ny > 0) {
            continue;
        }

        numWords++;
    }

    return numWords;
}


/*
  In this function the dictionary is reduced to the words that follows the pattern of word and colors given
*/
void reduceDictionary(char *word, char *colors) {
    char *auxDict = NULL;
    int auxTamDict = 0;

    for (int i = 0; i < tamDictionary; i++) {
        int numYellow = 0;
        char yellows[sizeWords];
        int numRed = 0;
        char red[sizeWords];
        int possible = 0;
        char validate[sizeWords];
        int pos;

        for (int j = 0; j < sizeWords; j++) {
            pos = i*sizeWords + j;

            for (int x = 0; x < numRed; x++) {
                if (red[x] == dictionary[pos]) {
                    break;
                }
            }

            if (colors[j] == 'G') {
                if (word[j] != dictionary[pos]) {
                    break;
                }
                else {
                    validate[j] = 'Y';
                }
            }
            else if (colors[j] == 'Y') {
                if (word[j] == dictionary[pos]) {
                    break;
                }
                else {
                    yellows[numYellow] = word[j];

                    numYellow++;
                    validate[j] = 'N';
                }
            }
            else {
                if (word[j] == dictionary[pos]) {
                    break;
                }
                else {
                    red[numRed] = word[j];

                    numRed++;
                    validate[j] = 'N';
                }
            }

            possible++;
        }

        if (possible < sizeWords) {
            continue;
        }

        possible = 1;

        int ny = numYellow;
        for (int x = 0; x < sizeWords; x++) {
            for (int y = 0; y < numYellow; y++) {
                if (yellows[y] == dictionary[i*sizeWords + x]) {
                    yellows[y] = 0;
                    ny--;
                    validate[x] = 'Y';
                    break;
                }
            }

            if (validate[x] == 'N') {
                for (int y = 0; y < numRed; y++) {
                    if (red[y] == dictionary[i*sizeWords + x]) {
                        possible = 0;
                        break;
                    }
                }
            }

            if (!possible) {
                break;
            }
        }

        if (!possible || ny > 0) {
            continue;
        }

        if (auxTamDict % 32 == 0) {
            if ((auxDict = (char *) realloc(auxDict, (auxTamDict + 32)*sizeWords)) == NULL) {
                fprintf(stderr, "Dynamic memory couldn't be allocated\n");
                exit(ERROR_DYNAMIC_MEMORY_ALLOCATION);
            }
        }

        strncpy(auxDict + auxTamDict*sizeWords, dictionary + i*sizeWords, sizeWords);
        auxTamDict++;
    }

    if ((dictionary = (char *) malloc(sizeWords*auxTamDict)) == NULL) {
        fprintf(stderr, "Dynamic memory couldn't be allocated\n");
        exit(ERROR_DYNAMIC_MEMORY_ALLOCATION);
    }

    memcpy(dictionary, auxDict, sizeWords*auxTamDict);
    tamDictionary = auxTamDict;

    free(auxDict);
}


/*
  This function is needed for the qsort function call
*/
int compEntropies(const void *ent1, const void *ent2) {
    if (((entropyType *)(ent1))->entropy - ((entropyType *)(ent2))->entropy > 0) {
        return -1;
    }
    else {
        return 1;
    }
}